#include "hip/hip_runtime.h"
__device__ int score_fun8(
    float xa[][3],
    float ya[][3],
    int n_ali,
    float d,
    int i_ali[],
    float *score1,
    int score_sum_method,
    float u[3][3],
    float t[3]
)
{
    float d0=dd0[blockIdx.x];
    float score_d8=dscore_d8[blockIdx.x];
    int Lnorm=dLnorm[blockIdx.x];

    float score_sum=0, di;
    float d_tmp=d*d;
    float d02=d0*d0;
    float score_d8_cut = score_d8*score_d8;
    int i, n_cut, inc=0;

    while(1)
    {
        n_cut=0;
        score_sum=0;
        for(i=0; i<n_ali; i++)
        {
            float xt[3];
            transform(t, u, xa[i], xt);
            di = dist(xt, ya[i]);
            if(di<d_tmp)
            {
                i_ali[n_cut]=i;
                n_cut++;
            }
            if(score_sum_method==8)
            {
                if(di<=score_d8_cut)
                {
                    score_sum += 1/(1+di/d02);
                }
            }
            else
            {
                score_sum += 1/(1+di/d02);
            }
        }
        //there are not enough feasible pairs, reliefe the threshold
        if(n_cut<3 && n_ali>3)
        {
            inc++;
            float dinc=(d+inc*0.5);//------做了修改,原来值是 0.5
            d_tmp = dinc * dinc;
        }
        else
        {
            break;
        }
    }

    *score1=score_sum/Lnorm;
    //if(blockIdx.x==1)
    //	cuPrintf("Lnorm  %d  score %f \n",Lnorm,*score1);
    return n_cut;
}


__device__ float TMscore8_search(
    float xtm[][3],
    float ytm[][3],
    int Lali,
    float t1[3],
    float u1[3][3],
    int simplify_step,
    int score_sum_method,
    int F
)
{
    int i, m;
    float d0_search=dd0_search[blockIdx.x];
    float  score=-1.0,score2=-1.0, rmsd;
    int k_ali[l1], ka, k;
    float d;
    float r1[l1][3],r2[l1][3],u[3][3],t[3];
    float u0[3][3];
    float t0[3];

    const int tid=threadIdx.y*blockDim.x+threadIdx.x;

    int n_it=20;
    const int n_init_max=6;//原来6
    int L_ini[n_init_max];
    int L_ini_min=4;
    if(Lali<4) L_ini_min=Lali;
    int n_init=0, i_init;
    for(i=0; i<n_init_max-1; i++)
    {
        n_init++;
        L_ini[i]=(int) (Lali/powf(2, (float) i));
        if(L_ini[i]<=L_ini_min)
        {
            L_ini[i]=L_ini_min;
            break;
        }
    }
    if(i==n_init_max-1)
    {
        n_init++;
        L_ini[i]=L_ini_min;
        //if(tid==0)
        //	cuPrintf("n_init %d Lali %d i %d\n",n_init,Lali,i);
    }

    int i_ali[l1], n_cut;
    int L_frag; //fragment length
    int iL_max; //maximum starting postion for the fragment
    i_init=threadIdx.y;
    i=0;
    if(i_init<n_init)
    //for(; i_init<n_init; i_init=i_init+blockDim.y)
    {
        L_frag=L_ini[i_init];
        iL_max=Lali-L_frag;
        i=threadIdx.x*simplify_step;
        while(1)
        {
            if(Lali-i<L_frag)
            {
                break;
            }

            //cuPrintf("======= %d %d \n",i,i_init);
            ka=0;

            for(k=0; k<L_frag; k++)
            {
                int kk=k+i;
                r1[k][0]=xtm[kk][0];
                r1[k][1]=xtm[kk][1];
                r1[k][2]=xtm[kk][2];

                r2[k][0]=ytm[kk][0];
                r2[k][1]=ytm[kk][1];
                r2[k][2]=ytm[kk][2];

                k_ali[ka]=kk;
                ka++;
            }

            Kabsch(r1, r2, L_frag, 1, &rmsd, t, u);
            d=d0_search-1;
            n_cut=score_fun8(xtm, ytm, Lali, d, i_ali, &score, score_sum_method,u,t);

            if(score>score2)
            {
                score2=score;
                for(k=0; k<3; k++)
                {
                    t0[k]=t[k];
                    u0[k][0]=u[k][0];
                    u0[k][1]=u[k][1];
                    u0[k][2]=u[k][2];
                }
            }

            d=d0_search+1;
            int it=0;
            for(; it<n_it; it++)
            {
                ka=0;
                for(k=0; k<n_cut; k++)
                {
                    m=i_ali[k];
                    r1[k][0]=xtm[m][0];
                    r1[k][1]=xtm[m][1];
                    r1[k][2]=xtm[m][2];

                    r2[k][0]=ytm[m][0];
                    r2[k][1]=ytm[m][1];
                    r2[k][2]=ytm[m][2];

                    k_ali[ka]=m;
                    ka++;
                }
                Kabsch(r1, r2, n_cut, 1, &rmsd, t, u);

                n_cut=score_fun8(xtm, ytm, Lali, d, i_ali, &score, score_sum_method,u,t);
                if(score>score2)
                {
                    score2=score;
                    for(k=0; k<3; k++)
                    {
                        t0[k]=t[k];
                        u0[k][0]=u[k][0];
                        u0[k][1]=u[k][1];
                        u0[k][2]=u[k][2];
                    }
                }

                if(n_cut==ka)
                {
                    for(k=0; k<n_cut; k++)
                    {
                        if(i_ali[k]!=k_ali[k])
                        {
                            break;
                        }
                    }
                    if(k==n_cut)
                    {
                        break; //stop iteration
                    }
                }
            } //for iteration
            if(i<iL_max)
            {
                i=i+simplify_step*blockDim.x;
                if(i>=iL_max) break; //i=iL_max;
            }
            else if(i>=iL_max)
                break;

        }//while(1)
    }//if(n_init)
    /**************************************************************/
    //cuPrintf("%f %d %d %d  Lali %d\n",score2,threadIdx.x,threadIdx.y,n_init,Lali);
    //if(F==1&&blockIdx.x==13)
    //cuPrintf("T: %f  %f  %f %d %d  %f  i:%d %d\n",t0[0],t0[1],t0[2],
    //threadIdx.x,threadIdx.y,score2,(i>=160? i-160:i),Lali);
    /*cuPrintf("%d %d %f  %f  %f %f\n",
    		threadIdx.x,
    		threadIdx.y,
    		t0[0],t0[1],t0[2],
    		score2);
    */
    /*************************************************************/

    __syncthreads();
    volatile __shared__ float sscore[32];
    volatile __shared__ int sscore_i[32];
    sscore_i[tid]=tid;
    sscore[tid]=score2;
    if(tid<16)
    {
        if(sscore[tid]<sscore[tid+16])
        {
            sscore[tid]=sscore[tid+16];
            sscore_i[tid]=sscore_i[tid+16];
        }

    }

    if(tid<8)
    {
        if(sscore[tid]<sscore[tid+8])
        {
            sscore[tid]=sscore[tid+8];
            sscore_i[tid]=sscore_i[tid+8];
        }
    }
    if(tid<4)
    {
        if(sscore[tid]<sscore[tid+4])
        {
            sscore[tid]=sscore[tid+4];
            sscore_i[tid]=sscore_i[tid+4];
        }
    }
    if(tid<2)
    {
        if(sscore[tid]<sscore[tid+2])
        {
            sscore[tid]=sscore[tid+2];
            sscore_i[tid]=sscore_i[tid+2];
        }
    }
    if(tid<1)
    {
        if(sscore[tid]<sscore[tid+1])
        {
            sscore[tid]=sscore[tid+1];
            sscore_i[tid]=sscore_i[tid+1];
        }
    }
    if(tid==sscore_i[0])
    {
        int k= 0;
        for(k=0; k<3; k++)
        {   t1[k]=t0[k];
            u1[k][0]=u0[k][0];
            u1[k][1]=u0[k][1];
            u1[k][2]=u0[k][2];

        }
        //cuPrintf("%f %f %f \n",	t1[0],t1[1],t1[2]);

        // printf(" %f  %d\n",sscore[0],Lali);
        //dtmscore[blockIdx.x]=sscore[0];
    }

    return sscore[0];
}

__device__ void Inster_TMscore8_search(
    float x[][3],
    float y[][3],
    int xlen,
    int ylen,
    int map[],
    int simplify_step,
    int score_sum_method,
    float *s
)
{
    int j=0;
    int k=0;
    const int tid=threadIdx.y*blockDim.x+threadIdx.x;
    volatile __shared__ int sk;
    if(tid<3)
    {
        for(j=0; j<ylen; j++)
        {
            int i=map[j];
            if(i>=0&&i<xlen)
            {
                xtm1[blockIdx.x][k][tid]=x[i][tid];
                ytm1[blockIdx.x][k][tid]=y[j][tid];
                k++;

                //		if(tid==0)
                //			cuPrintf("%d -> %d \n",j,i);
            }
            if(tid==0)
                sk=k;
        }
    }

    float TM=TMscore8_search(
                 xtm1[blockIdx.x],
                 ytm1[blockIdx.x],
                 sk,
                 t1[blockIdx.x],
                 u1[blockIdx.x],
                 simplify_step,
                 score_sum_method,
                 0);



    if(tid==0)
    {
        /*
        	cuPrintf("%f %f %f \n",
        		t1[blockIdx.x][0],
        		t1[blockIdx.x][1],
        		t1[blockIdx.x][2]);
        */
        //dtmscore[blockIdx.x]=TM;
        dtmscore2[blockIdx.x]=TM;
        //cuPrintf("TM == %f \n",TM);
    }
    __syncthreads();
}


__global__ void detailed_search(
    float x [][3],
    float y[][3],
    int xlen,
    int ylen[],
    int simplify_step,
    int score_sum_method,
    const int l22,
    float score[])
{
    Inster_TMscore8_search(
        x,
        &y[blockIdx.x*l22],
        xlen,
        ylen[blockIdx.x],
        invmap[blockIdx.x],
        simplify_step,
        score_sum_method,
        NULL);


}


__global__ void detailed_search1(
    float x [][3],
    float y[][3],
    int xlen,
    int ylen[],
    int simplify_step,
    int score_sum_method,
    const int l22,
    float *s)
{
    Inster_TMscore8_search(
        x,
        &y[blockIdx.x*l22],
        xlen,
        ylen[blockIdx.x],
        invmap[blockIdx.x],
        simplify_step,
        score_sum_method,
        NULL);


    const int tid=threadIdx.y*blockDim.x+threadIdx.x;
    if(tid==0)
        dtmscore[blockIdx.x]=dtmscore2[blockIdx.x];

    for(int j=tid; j<l22; j=j+32)
    {
        invmapbak[blockIdx.x][j]=invmap[blockIdx.x][j];
        //s[blockIdx.x*l2+j]=invmap2[blockIdx.x][8][j];
    }
}


__global__ void detailed_search2(
    float x [][3],
    float y[][3],
    int xlen,
    int ylen[],
    int simplify_step,
    int score_sum_method,
    const int l22,
    float *s)
{
    Inster_TMscore8_search(
        x,
        &y[blockIdx.x*l22],
        xlen,
        ylen[blockIdx.x],
        invmap[blockIdx.x],
        simplify_step,
        score_sum_method,
        NULL);


    const int tid=threadIdx.y*blockDim.x+threadIdx.x;
    volatile __shared__  int flag;
    if(tid==0)
    {   flag=0;
        if(dtmscore[blockIdx.x]<dtmscore2[blockIdx.x])
        {
            dtmscore[blockIdx.x]=dtmscore2[blockIdx.x];
            flag=1;
        }
    }

    if(flag==1)
    {
        for(int i=tid; i<l22; i=i+32)
        {
            invmapbak[blockIdx.x][i]=invmap[blockIdx.x][i];
        }
    }

    //for(int i=tid;i<l22;i=i+32)
    //	         s[blockIdx.x*l2+i]=invmapbak[blockIdx.x][i];

}

__global__ void detailed_search3(
    float x [][3],
    float y[][3],
    int xlen,
    int ylen[],
    int simplify_step,
    int score_sum_method,
    const int l22,
    float *s)
{
    Inster_TMscore8_search(
        x,
        &y[blockIdx.x*l22],
        xlen,
        ylen[blockIdx.x],
        invmap[blockIdx.x],
        simplify_step,
        score_sum_method,
        NULL);


    const int tid=threadIdx.y*blockDim.x+threadIdx.x;
    volatile __shared__  int flag;
    if(tid==0)
    {   flag=0;
        if(dtmscore[blockIdx.x]<dtmscore2[blockIdx.x])
        {
            dtmscore[blockIdx.x]=dtmscore2[blockIdx.x];
            flag=1;
        }
    }

    if(flag==1)
    {
        for(int i=tid; i<l22; i=i+32)
        {
            invmapbak[blockIdx.x][i]=invmap[blockIdx.x][i];
        }
    }

}

__global__ void detailed_search4(
    float x [][3],
    float y[][3],
    int xlen,
    int ylen[],
    int simplify_step,
    int score_sum_method,
    const int l22,
    float *s)
{
    Inster_TMscore8_search(
        x,
        &y[blockIdx.x*l22],
        xlen,
        ylen[blockIdx.x],
        invmap[blockIdx.x],
        simplify_step,
        score_sum_method,
        NULL);


    const int tid=threadIdx.y*blockDim.x+threadIdx.x;
    volatile __shared__  int flag;
    if(tid==0)
    {   flag=0;
        if(dtmscore[blockIdx.x]<dtmscore2[blockIdx.x])
        {
            dtmscore[blockIdx.x]=dtmscore2[blockIdx.x];
            flag=1;
        }
    }

    if(flag==1)
    {

        for(int i=tid; i<l22; i=i+32)
        {

            invmapbak[blockIdx.x][i]=invmap[blockIdx.x][i];
        }
    }

}

__global__ void detailed_search5(
    float x [][3],
    float y[][3],
    int xlen,
    int ylen[],
    int simplify_step,
    int score_sum_method,
    const int l22,
    float *s)
{
    Inster_TMscore8_search(
        x,
        &y[blockIdx.x*l2],
        xlen,
        ylen[blockIdx.x],
        invmap[blockIdx.x],
        simplify_step,
        score_sum_method,
        NULL);


    const int tid=threadIdx.y*blockDim.x+threadIdx.x;
    volatile __shared__  int flag;
    if(tid==0)
    {   flag=0;
        if(dtmscore[blockIdx.x]<dtmscore2[blockIdx.x])
        {
            dtmscore[blockIdx.x]=dtmscore2[blockIdx.x];
            flag=1;
        }
    }

    if(flag==1)
    {

        for(int i=tid; i<l22; i=i+32)
        {

            invmapbak[blockIdx.x][i]=invmap[blockIdx.x][i];
        }
    }
    //for(int i=tid;i<l22;i=i+32)
    //	         s[blockIdx.x*l2+i]=invmapbak[blockIdx.x][i];

}


__global__ void detailed_search6(
    float x [][3],
    float y[][3],
    int xlen,
    int ylen[],
    int simplify_step,
    int score_sum_method,
    const int l22,
    float score[])
{
    Inster_TMscore8_search(
        x,
        &y[blockIdx.x*l22],
        xlen,
        ylen[blockIdx.x],
        invmapbak[blockIdx.x],
        simplify_step,
        score_sum_method,
        NULL);


}
