#include "hip/hip_runtime.h"

__device__ float get_initial_local(
    float x[][3],
    float y[][3],
    int x_len,
    int y_len,
    int *y2x,
    const int l,
    float val[],
    char path[])
{
    float GL, rmsd;
    float d01=dd0[blockIdx.x]+1.5;
    if(d01 < dD0_MIN[blockIdx.x]) d01=dD0_MIN[blockIdx.x];
    float d02=d01*d01;

    float GLmax=0;
    int n_frag=20;
    int ns=20;

    int aL=(x_len<=y_len?x_len:y_len);
    if(aL>250)
    {
        n_frag=50;
    }
    else if(aL>200)
    {
        n_frag=40;
    }
    else if(aL>150)
    {
        n_frag=30;
    }
    else
    {
        n_frag=20;
    }

    int smallest=aL/3;

    if(n_frag>smallest) n_frag=smallest;
    if(ns>smallest) ns=smallest;

    int m1=x_len-n_frag-ns;
    int m2=y_len-n_frag-ns;
    int i,j,k;
    const int tid=threadIdx.y*blockDim.x+threadIdx.x;

    float r1[32][3],r2[32][3],u[3][3],t[3];
    __shared__ float t0[32][3],u0[32][3][3];
//	volatile __shared__ int sinvmap[l2];

    volatile __shared__ int sk;
    volatile __shared__ float sGLmax[32];
    sGLmax[tid]=-1;
    i=ns-1;
    for(; i<m1; i=i+n_frag)
    {

        j=ns-1+tid*n_frag;
        if(j<m2)
        {
            for(k=0; k<n_frag; k++)
            {
                r1[k][0]=x[k+i][0];
                r1[k][1]=x[k+i][1];
                r1[k][2]=x[k+i][2];

                r2[k][0]=y[k+j][0];
                r2[k][1]=y[k+j][1];
                r2[k][2]=y[k+j][2];
            }
            Kabsch(r1, r2, n_frag, 1, &rmsd, t, u);
            for(k=0; k<3; k++)
            {
                t0[tid][k]=t[k];
                u0[tid][k][0]=u[k][0];
                u0[tid][k][1]=u[k][1];
                u0[tid][k][2]=u[k][2];
            }

            //printf(" %d %f %f %f\n",tid,t0[tid][0],t0[tid][1],t0[tid][2]);
        }
        /*

        		k=0;
        		for(j=ns-1; j<m2; j=j+n_frag)
        		{
        			float gap_open=0.0;
        			DNW(x, y,
        				x_len,
        				y_len,
                        		t0[k],
                        		u0[k],
                        		d02,
                        		gap_open,
                        		invmap[blockIdx.x],
                       			val, path);
        			k++;
        			if(tid<k)
        			{
        				GL=get_score_fast(
        					x,
        					y,
        				 	x_len,
        					y_len,
        					invmap[blockIdx.x]);
        					//printf("%d %f \n",tid,GL);
        				if(GL>GLmax)
        				{
        					GLmax=GL;

        					for(int k1=tid;k1<l2;k1=k1+32)
        					sinvmap[k1]=invmap[blockIdx.x][k1];

        				}
        			}
        		}

        */

        k=0;
        for(j=ns-1; j<m2; j=j+n_frag)
        {
            float gap_open=0.0;
            DNW(x, y,
                x_len,
                y_len,
                t0[k],
                u0[k],
                d02,
                gap_open,
                invmap2[blockIdx.x][k],
		l,
                val, path);
            k++;
        }

        if(tid<k)
        {
            GL=get_score_fast(
                   x,
                   y,
                   x_len,
                   y_len,
                   invmap2[blockIdx.x][tid]);
            //printf("%f\n",GL);
            if(GL>sGLmax[tid])
            {
                sGLmax[tid]=GL;
            }
        }

        if( tid==0)
        {
            int i;
            GLmax=0;
            for(i=0; i<8; i++)
                if(GLmax<sGLmax[i])
                {
                    GLmax=sGLmax[i];
                    sk=i;
                }
        }

        if(sGLmax[9]<sGLmax[sk])
        {
            for(k=tid; k<l; k=k+32)
                invmap[blockIdx.x][k]=invmap2[blockIdx.x][sk][k];
            if(tid==0)
                sGLmax[9]=sGLmax[sk];
        }
    }
    GLmax=sGLmax[9];
    //if(threadIdx.x==0&&threadIdx.y==0)
    //	printf("%f \n",GLmax);
    return GLmax;
}

__global__ void get_initial_local2(
    float x[][3],
    float y[][3],
    int x_len,
    int y_len[],
    const int l22,
    float val[],
    char path[],
    float *s)
{
    //float GL=
    get_initial_local(
        x,
        &y[blockIdx.x*l22],
        x_len,
        y_len[blockIdx.x],
        invmap[blockIdx.x],l22,
        &val[blockIdx.x*(x_len+1)*(l22+1)],
        &path[blockIdx.x*(x_len+1)*(l22+1)]);
    /*
    	const int tid=threadIdx.y*blockDim.x+threadIdx.x;
    	//if(threadIdx.x==0&&threadIdx.y==0)
    	//	s[blockIdx.x]=GL;
    	for(int k=tid;k<l2;k=k+32)
    		s[blockIdx.x*l2+k]=invmap[blockIdx.x][k];

    */
}
