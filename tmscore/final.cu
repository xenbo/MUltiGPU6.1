#include "hip/hip_runtime.h"

__device__ void parameter_set4final(float len)
{
    float D0_MIN=0.5;
    float d0;
    int Lnorm=len;
    if(Lnorm<=21)
    {
        d0=0.5;
    }
    else
    {
        d0=(1.24*pow((Lnorm*1.0-15), 1.0/3)-1.8);
    }
    if(d0<D0_MIN) d0=D0_MIN;

    float d0_search=d0;
    if(d0_search>8) d0_search=8;
    if(d0_search<4.5) d0_search=4.5;


    dLnorm[blockIdx.x]=Lnorm;
    dd0[blockIdx.x]=d0;
    dD0_MIN[blockIdx.x]=D0_MIN;
    dd0_search[blockIdx.x]=d0_search;
}

__device__ void final_TMscore8_search(
    float x[][3],
    float y[][3],
    int xlen,
    int ylen,
    int map[],
    float *s)
{
    int j=0, k=0;
    float d;
    const int tid=threadIdx.y*blockDim.x+threadIdx.x;
    float  score_d8= dscore_d8[blockIdx.x];
    volatile __shared__ int sk;

    if(tid<3)
    {
        for(j=0; j<ylen; j++)
        {
            int i=map[j];
            if(i>=0&&i<xlen)
            {
                float xt[3];
                transform(t1[blockIdx.x], u1[blockIdx.x], x[i], xt);

                float d0=xt[0]-y[j][0];
                float d1=xt[1]-y[j][1];
                float d2=xt[2]-y[j][2];

                d=d0*d0+d1*d1+d2*d2;

                if(d <= score_d8*score_d8)
                {
                    xtm1[blockIdx.x][k][tid]=x[i][tid];
                    ytm1[blockIdx.x][k][tid]=y[j][tid];
                    k++;
                }

            }
            if(tid==0)
                sk=k;
        }
    }

    if(tid==0)
        parameter_set4final(ylen+0.0);

    float TM1=TMscore8_search3(
                  xtm1[blockIdx.x],
                  ytm1[blockIdx.x],
                  sk,
                  t1[blockIdx.x],
                  u1[blockIdx.x],
                  1,0,0);
    if(tid==0)
        parameter_set4final(xlen+0.0);

    float TM2=TMscore8_search3(
                  xtm1[blockIdx.x],
                  ytm1[blockIdx.x],
                  sk,
                  t1[blockIdx.x],
                  u1[blockIdx.x],
                  1,0,0);
    if(tid==0)
    {

        dtmscore[blockIdx.x]=TM1;
        dtmscore2[blockIdx.x]=TM2;
    }
    __syncthreads();
}



__global__ void Gfinal_TMscore8_search(
    float x[][3],
    float y[][3],
    int xlen,
    int ylen[],
    const int l22,
    float score[])
{
    //if(blockIdx.x==5)
    final_TMscore8_search(
        x,
        &y[blockIdx.x*l22],
        xlen,
        ylen[blockIdx.x],
        invmapbak[blockIdx.x],
        NULL);


    //const int tid=threadIdx.y*blockDim.x+threadIdx.x;
    /*
    if(tid==0&&blockIdx.x==0)
    for(int j=0;j<ylen[blockIdx.x];j++)
    {
    		int i=invmap[blockIdx.x][j];
    		if(i>=0&&i<xlen)
    		{
    			cuPrintf("%d -> %d \n",j,i);
    		}

    }
    */
}

