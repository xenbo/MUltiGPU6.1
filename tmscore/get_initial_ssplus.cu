#include "hip/hip_runtime.h"
__device__ void NWDP_TM3(
    int len1,
    int len2,
    float gap_open,
    int j2i[],
    float score[],
    const int l,
    float val[],
    char path[])
{

    int i, j;
    float h, v, d;
    const int tid=threadIdx.y*blockDim.x+threadIdx.x;
    for(i=tid; i<=len1; i+=32)
    {
        val[i*(l+1)+0]=0;
        path[i*(l+1)+0]=0;
    }

    for(j=tid; j<=len2; j+=32)
    {
        val[j]=0;
        path[j]=0;
        j2i[j]=-1;
    }

    int nv=1;
    i=tid+1;
    j=1;


    while(i<=len1)
    {
        if(i<=nv)
        {
            if(j<=len2)
            {
                d=val[(i-1)*(l+1)+j-1]+score[i*(l+1)+j];

                h=val[(i-1)*(l+1)+j];
                if(path[(i-1)*(l+1)+j])
                    h += gap_open;

                v=val[i*(l+1)+j-1];
                if(path[i*(l+1)+j-1])
                    v += gap_open;


                if(d>=h && d>=v)
                {
                    path[i*(l+1)+j]=1;
                    val[i*(l+1)+j]=d;
                }
                else
                {
                    path[i*(l+1)+j]=0;
                    if(v>=h)
                        val[i*(l+1)+j]=v;
                    else
                        val[i*(l+1)+j]=h;
                }
                j++;
            }
        }
        nv++;
        if(j>len2)
        {
            i+=32;
            j=1;
        }
    }
    if(tid==0)
    {
        i=len1;
        j=len2;
        while(i>0 && j>0)
        {
            if(path[i*(l+1)+j])
            {
                j2i[j-1]=i-1;
                i--;
                j--;
            }
            else
            {
                h=val[(i-1)*(l+1)+j];
                if(path[(i-1)*(l+1)+j]) h +=gap_open;

                v=val[i*(l+1)+j-1];
                if(path[i*(l+1)+j-1]) v +=gap_open;

                if(v>=h)
                    j--;
                else
                    i--;
            }
        }
    }
}

__device__ void score_matrix_rmsd_sec(
    float x[][3],
    float y[][3],
    int x_len,
    int y_len,
    int secx[],
    int secy[],
    const int l,
    int *y2x,
    float score[])
{
    const int tid=threadIdx.y*blockDim.x+threadIdx.x;
    __shared__ float t[3], u[3][3];
    float rmsd, dij;
    float d01=dd0[blockIdx.x]+1.5;
    if(d01 < dD0_MIN[blockIdx.x]) d01=dD0_MIN[blockIdx.x];
    float d02=d01*d01;

    float xx[3];
    if(tid==0)
    {
        int i, k=0;
        float r1[l1][3],r2[l1][3];
        for(int j=0; j<y_len; j++)
        {
            i=y2x[j];
            if(i>=0&&i<x_len)
            {
                r1[k][0]=x[i][0];
                r1[k][1]=x[i][1];
                r1[k][2]=x[i][2];

                r2[k][0]=y[j][0];
                r2[k][1]=y[j][1];
                r2[k][2]=y[j][2];

                k++;
            }
        }
        Kabsch(r1, r2, k, 1, &rmsd, t, u);
    }
    for(int ii=threadIdx.x; ii<x_len; ii+=blockDim.x)
    {
        transform(t, u, &x[ii][0], xx);
        for(int jj=threadIdx.y; jj<y_len; jj+=blockDim.y)
        {
            dij=dist(xx, &y[jj][0]);
            if(secx[ii]==secy[jj])
            {
                score[(ii+1)*(l+1)+jj+1] = 1.0/(1+dij/d02) + 0.5;
            }
            else
            {
                score[(ii+1)*(l+1)+jj+1] = 1.0/(1+dij/d02);
            }
        }
    }
}


__device__ void get_initial_ssplus(
    float x[][3],
    float y[][3],
    int x_len,
    int y_len,
    int secx[],
    int secy[],
    int *y2xb,
    int *y2x,
    const int l,
    float score[],
    float val[],
    char path[])
{

    score_matrix_rmsd_sec(x, y, x_len, y_len,secx,secy,l,y2xb,score);
    float gap_open=-1.0;
    NWDP_TM3(x_len, y_len, gap_open, y2x,score,l,val,path);
}

__global__ void get_initial_ssplus2(
    float x[][3],
    float y[][3],
    int x_len,
    int y_len[],
    const int l22,
    float score[],
    float val[],
    char path[],
    float *s)
{

    get_initial_ssplus( x,
                        &y[blockIdx.x*l22],
                        x_len,
                        y_len[blockIdx.x],
                        secx[blockIdx.x],
                        secy[blockIdx.x],
                        invmapbak[blockIdx.x],
                        invmap[blockIdx.x],l22,
                        &score[blockIdx.x*(x_len+1)*(l22+1)],
                        &val[blockIdx.x*(x_len+1)*(l22+1)],
                        &path[blockIdx.x*(x_len+1)*(l22+1)]
                      );


    /*
    	const int tid=threadIdx.y*blockDim.x+threadIdx.x;
    	for(int i=tid;i<l2;i=i+32)
    	{
    		s[blockIdx.x*l2+i]=invmap[blockIdx.x][i];
    	}
    */
}

