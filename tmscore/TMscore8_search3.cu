#include "hip/hip_runtime.h"

__device__ float TMscore8_search3(
    float xtm[][3],
    float ytm[][3],
    int Lali,
    float t1[3],
    float u1[3][3],
    int simplify_step,
    int score_sum_method,
    int F
)
{
    int i, m;
    float d0_search=dd0_search[blockIdx.x];
    float  score=-1.0,score2=-1.0, rmsd;
    int k_ali[l1], ka, k;
    float d;
    float u[3][3],t[3];
    float u0[3][3];
    float t0[3];
	int i_ali[l1], n_cut;
    const int tid=threadIdx.y*blockDim.x+threadIdx.x;

    const int n_it=20;
    const int n_init_max=6;
    int L_ini[n_init_max];
    int L_ini_min=4;
    if(Lali<4) L_ini_min=Lali;
    int n_init=0;
    for(i=0; i<n_init_max-1; i++)
    {
        n_init++;
        L_ini[i]=(int) (Lali/__powf(2.0, (float) i));
          
        if(L_ini[i]<=L_ini_min)
        {
            L_ini[i]=L_ini_min;
            break;
        }
    }
    if(i==n_init_max-1)
    {
        n_init++;
        L_ini[i]=L_ini_min;
    }


    const int nu[]= {1,3,5,7,9,7};
    const int hh=(int)__powf(tid,(float)0.5);

    if(hh<n_init)
    {
        //printf("%d  %d  %d\n",tid,hh,L_ini[hh]);

        const int s=tid-hh*hh;
        for(int jid=s; jid<=Lali-L_ini[hh]; jid=jid+nu[hh])
        {

            //printf("%d-%d \n",jid,jid+L_ini[hh]);

            k=0;
            float r1[l1][3],r2[l1][3];
            for(int kk=jid; kk<jid+L_ini[hh]; kk++)
            {

                r1[k][0]=xtm[kk][0];
                r1[k][1]=xtm[kk][1];
                r1[k][2]=xtm[kk][2];

                r2[k][0]=ytm[kk][0];
                r2[k][1]=ytm[kk][1];
                r2[k][2]=ytm[kk][2];
                k++;
            }

            Kabsch(r1, r2, k, 1, &rmsd, t, u);
            d=d0_search-1;
            n_cut=score_fun8(xtm, ytm, Lali, d, i_ali, &score, score_sum_method,u,t);

            if(score>score2)
            {
                score2=score;
                for(k=0; k<3; k++)
                {
                    t0[k]=t[k];
                    u0[k][0]=u[k][0];
                    u0[k][1]=u[k][1];
                    u0[k][2]=u[k][2];
                }
            }

            d=d0_search+1;
            int it=0;
            for(; it<n_it; it++)
            {
                ka=0;
                for(k=0; k<n_cut; k++)
                {
                    m=i_ali[k];
                    r1[k][0]=xtm[m][0];
                    r1[k][1]=xtm[m][1];
                    r1[k][2]=xtm[m][2];

                    r2[k][0]=ytm[m][0];
                    r2[k][1]=ytm[m][1];
                    r2[k][2]=ytm[m][2];

                    k_ali[ka]=m;
                    ka++;
                }
                Kabsch(r1, r2, n_cut, 1, &rmsd, t, u);

                n_cut=score_fun8(xtm, ytm, Lali, d, i_ali, &score, score_sum_method,u,t);
                if(score>score2)
                {
                    score2=score;
                    for(k=0; k<3; k++)
                    {
                        t0[k]=t[k];
                        u0[k][0]=u[k][0];
                        u0[k][1]=u[k][1];
                        u0[k][2]=u[k][2];
                    }
                }

                if(n_cut==ka)
                {
                    for(k=0; k<n_cut; k++)
                    {
                        if(i_ali[k]!=k_ali[k])
                        {
                            break;
                        }
                    }
                    if(k==n_cut)
                    {
                        break; //stop iteration
                    }
                }
            } //for iteration
        }
    }
    __syncthreads();
    volatile __shared__ float sscore[32];
    volatile __shared__ int sscore_i[32];
    sscore_i[tid]=tid;
    sscore[tid]=score2;
    if(tid<16)
    {
        if(sscore[tid]<sscore[tid+16])
        {
            sscore[tid]=sscore[tid+16];
            sscore_i[tid]=sscore_i[tid+16];
        }

    }

    if(tid<8)
    {
        if(sscore[tid]<sscore[tid+8])
        {
            sscore[tid]=sscore[tid+8];
            sscore_i[tid]=sscore_i[tid+8];
        }
    }
    if(tid<4)
    {
        if(sscore[tid]<sscore[tid+4])
        {
            sscore[tid]=sscore[tid+4];
            sscore_i[tid]=sscore_i[tid+4];
        }
    }
    if(tid<2)
    {
        if(sscore[tid]<sscore[tid+2])
        {
            sscore[tid]=sscore[tid+2];
            sscore_i[tid]=sscore_i[tid+2];
        }
    }
    if(tid<1)
    {
        if(sscore[tid]<sscore[tid+1])
        {
            sscore[tid]=sscore[tid+1];
            sscore_i[tid]=sscore_i[tid+1];
        }
    }
    if(tid==sscore_i[0])
    {
        int k= 0;
        for(k=0; k<3; k++)
        {   t1[k]=t0[k];
            u1[k][0]=u0[k][0];
            u1[k][1]=u0[k][1];
            u1[k][2]=u0[k][2];

        }
        //cuPrintf("%f %f %f \n",	t1[0],t1[1],t1[2]);

        //cuPrintf(" %f  %d\n",sscore[0],Lali);
        //dtmscore[blockIdx.x]=sscore[0];
    }

    return sscore[0];

}

__device__ void Inster_TMscore8_search3(
    float x[][3],
    float y[][3],
    int xlen,
    int ylen,
    int map[],
    int simplify_step,
    int score_sum_method,
    float *s
)
{
    int j=0;
    int k=0;
    const int tid=threadIdx.y*blockDim.x+threadIdx.x;
    volatile __shared__ int sk;
    if(tid<3)
    {
        for(j=0; j<ylen; j++)
        {
            int i=map[j];
            if(i>=0&&i<xlen)
            {
                xtm1[blockIdx.x][k][tid]=x[i][tid];
                ytm1[blockIdx.x][k][tid]=y[j][tid];
                k++;
            }
            if(tid==0)
                sk=k;
        }
    }

    float TM=TMscore8_search3(
                 xtm1[blockIdx.x],
                 ytm1[blockIdx.x],
                 sk,
                 t1[blockIdx.x],
                 u1[blockIdx.x],
                 simplify_step,
                 score_sum_method,
                 0);

	if(tid==0)
        dtmscore2[blockIdx.x]=TM;
   

}


__global__ void detailed_search6(
    float x [][3],
    float y[][3],
    int xlen,
    int ylen[],
    int simplify_step,
    int score_sum_method,
    const int l22,
    float score[])
{
    Inster_TMscore8_search3(
        x,
        &y[blockIdx.x*l22],
        xlen,
        ylen[blockIdx.x],
        invmapbak[blockIdx.x],
        simplify_step,
        score_sum_method,
        NULL);

}
