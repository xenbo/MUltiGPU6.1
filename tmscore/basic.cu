
#include <hip/hip_runtime.h>
__device__ float dist(float x[3], float y[3])
{
    float d1=x[0]-y[0];
    float d2=x[1]-y[1];
    float d3=x[2]-y[2];

    return (d1*d1 + d2*d2 + d3*d3);
}

__device__ float dot(float a[], float b[])
{
    return (a[0] * b[0] + a[1] * b[1] + a[2] * b[2]);
}

__device__ void transform(float t[3], float u[3][3], float *x, float *x1)
{
    x1[0]=t[0]+dot(&u[0][0], x);
    x1[1]=t[1]+dot(&u[1][0], x);
    x1[2]=t[2]+dot(&u[2][0], x);
}

__device__ void do_rotation(float x[][3], float x1[][3], int len, float t[3], float u[3][3])
{
    int i=0;
    for(; i<len; i++)
    {
        transform(t,u,&x[i][0],&x1[i][0]);
    }
}

char* get50(int a,char *c)
{
    c[0]=c[1]=c[2]=c[3]=c[4]='0';
    c[5]='\0';
    if(a<10)	{
        c[4]='\0';
        return c;
    }
    if(a<100)	{
        c[3]='\0';
        return c;
    }
    if(a<1000)	{
        c[2]='\0';
        return c;
    }
    if(a<10000)	{
        c[1]='\0';
        return c;
    }
    if(a<100000) {
        c[0]='\0';
        return c;
    }

    return c;
}
