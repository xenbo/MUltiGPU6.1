#include "hip/hip_runtime.h"
__device__ float get_score_fast(float x[][3],float y[][3],int x_len, int y_len, int map[])
{
    float rms, tmscore, tmscore1, tmscore2,r1[l1][3],r2[l1][3],xtm[l1][3],ytm[l1][3],u[3][3],t[3];
    int i, j, k;

    k=0;
    for(j=0; j<y_len; j++)
    {
        i=map[j];
        if(i>=0&&i<x_len)
        {
            /*
            r1[k][0]=x[i][0];
            r1[k][1]=x[i][1];
            r1[k][2]=x[i][2];

            r2[k][0]=y[j][0];
            r2[k][1]=y[j][1];
            r2[k][2]=y[j][2];
            */

            xtm[k][0]=x[i][0];
            xtm[k][1]=x[i][1];
            xtm[k][2]=x[i][2];

            ytm[k][0]=y[j][0];
            ytm[k][1]=y[j][1];
            ytm[k][2]=y[j][2];

            k++;
        }

    }
    Kabsch(xtm, ytm, k, 1, &rms, t, u);
    // cuPrintf("%f %f %f %f %f %f \n",r1[0][0],r1[0][1],r1[0][2],u[0][0],u[0][1],u[0][2]);
    float di;
    float dis[l1];
    float d0_search=dd0_search[blockIdx.x];
    float d00=d0_search;
    float d002=d00*d00;
    float d0=dd0[blockIdx.x];
    float d02=d0*d0;

    int n_ali=k;
    float xrot[3];
    tmscore=0;
    for(k=0; k<n_ali; k++)
    {
        transform(t, u, &xtm[k][0], xrot);
        di=dist(xrot, &ytm[k][0]);
        dis[k]=di;
        tmscore +=  1/(1+di/d02);
    }

    //second iteration
    float d002t=d002;
    while(1)
    {
        j=0;
        for(k=0; k<n_ali; k++)
        {
            if(dis[k]<=d002t)
            {
                r1[j][0]=xtm[k][0];
                r1[j][1]=xtm[k][1];
                r1[j][2]=xtm[k][2];

                r2[j][0]=ytm[k][0];
                r2[j][1]=ytm[k][1];
                r2[j][2]=ytm[k][2];

                j++;
            }
        }

        if(j<3 && n_ali>3)
        {
            d002t += 0.5;//------做了修改(默认0.5)
        }
        else
        {
            break;
        }
    }

    if(n_ali!=j)
    {
        Kabsch(r1, r2, j, 1, &rms, t, u);
        tmscore1=0;
        for(k=0; k<n_ali; k++)
        {
            transform(t, u, &xtm[k][0], xrot);
            di=dist(xrot, &ytm[k][0]);
            dis[k]=di;
            tmscore1 += 1/(1+di/d02);
        }

        d002t=d002+1;

        while(1)
        {
            j=0;
            for(k=0; k<n_ali; k++)
            {
                if(dis[k]<=d002t)
                {
                    r1[j][0]=xtm[k][0];
                    r1[j][1]=xtm[k][1];
                    r1[j][2]=xtm[k][2];

                    r2[j][0]=ytm[k][0];
                    r2[j][1]=ytm[k][1];
                    r2[j][2]=ytm[k][2];

                    j++;
                }
            }

            if(j<3 && n_ali>3)
            {
                d002t += 0.5;//------ 做了修改（默认0.5）
            }
            else
            {
                break;
            }
        }

        Kabsch(r1, r2, j, 1, &rms, t, u);
        tmscore2=0;
        for(k=0; k<n_ali; k++)
        {
            transform(t, u, &xtm[k][0], xrot);
            di=dist(xrot, &ytm[k][0]);
            tmscore2 += 1/(1+di/d02);
        }
    }
    else
    {
        tmscore1=tmscore;
        tmscore2=tmscore;
    }

    if(tmscore1>=tmscore) tmscore=tmscore1;
    if(tmscore2>=tmscore) tmscore=tmscore2;

    return tmscore;
}


__device__ void get_initial(
    float x[][3],
    float y[][3],
    int x_len,
    int y_len,
    int *y2x2,
    float *s)
{
    int min_len=(x_len<=y_len? x_len:y_len);
    int min_ali= min_len/2;
    if(min_ali<5) min_ali=5;

    int nn1= min_ali-y_len;
    int nn2=x_len-min_ali;

    int  y2x[l2];
    int i, j, k, k_best;
    float tmscore=-1, tmscore_max=-1;

    const int tid=threadIdx.y*blockDim.x+threadIdx.x;
    k_best=nn1+tid;
    for(k=nn1+tid; k<=nn2; k=k+blockDim.x)
    {
        for(j=0; j<y_len; j++)
        {
            i=j+k;
            if(i>=0 && i<x_len)
            {
                y2x[j]=i;
            }
            else
            {
                y2x[j]=-1;
            }
        }


        tmscore=get_score_fast(x, y, x_len, y_len, y2x);
        if(tmscore>=tmscore_max)
        {
            tmscore_max=tmscore;
            k_best=k;
        }
        //s[tid]=tmscore;
    }


    //合并归约
    volatile __shared__ float sscore[32];
    volatile __shared__  int  sscore_i[32];
    sscore_i[tid]=k_best;
    sscore[tid]=tmscore_max;

    if(tid<16)
    {
        if(sscore[tid]<sscore[tid+16])
        {
            sscore[tid]=sscore[tid+16];
            sscore_i[tid]=sscore_i[tid+16];
        }
    }

    if(tid<8)
    {
        if(sscore[tid]<sscore[tid+8])
        {
            sscore[tid]=sscore[tid+8];
            sscore_i[tid]=sscore_i[tid+8];
        }
    }
    if(tid<4)
    {
        if(sscore[tid]<sscore[tid+4])
        {
            sscore[tid]=sscore[tid+4];
            sscore_i[tid]=sscore_i[tid+4];
        }
    }
    if(tid<2)
    {
        if(sscore[tid]<sscore[tid+2])
        {
            sscore[tid]=sscore[tid+2];
            sscore_i[tid]=sscore_i[tid+2];
        }
    }
    if(tid<1)
    {
        if(sscore[tid]<sscore[tid+1])
        {
            sscore[tid]=sscore[tid+1];
            sscore_i[tid]=sscore_i[tid+1];
        }
        //printf("sscore %f k_best %d \n",sscore[0],sscore_i[0]);
        //*s=sscore[0];
    }

    k=sscore_i[0];
    for(j=tid; j<y_len; j=j+blockDim.x)
    {
        i=j+k;
        if(i>=0 && i<x_len)
        {
            y2x2[j]=i;
        }
        else
        {
            y2x2[j]=-1;
        }
    }
}
__global__ void get_initial2( float x[][3],
                              float y[][3],
                              int x_len,
                              int y_len[],
                              const int l22,
                              float *s
                            )
{
    //cuPrintf("== %d \n",y_len[blockIdx.x]);
    //if(blockIdx.x==14)
    get_initial(
        x,
        &y[blockIdx.x*l22],
        x_len,
        y_len[blockIdx.x],
        invmap[blockIdx.x],
        NULL);

}
