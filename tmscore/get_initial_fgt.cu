#include "hip/hip_runtime.h"
__device__  int find_max_frag(float x[][3],int resno[], int len)
{

    const int tid=threadIdx.y*blockIdx.x+threadIdx.x;
    unsigned short start_max,end_max;
    volatile __shared__ unsigned short start_max2;
    volatile __shared__ unsigned short end_max2;

    int r_min, fra_min=4;
    float d;
    int start;
    int Lfr_max=0, flag;

    r_min= (int) (len*1.0/3.0);
    if(r_min > fra_min) r_min=fra_min;

    int inc=0;
    float dcu0_cut=ddcu0[blockIdx.x]*ddcu0[blockIdx.x];
    float dcu_cut=dcu0_cut;

    //inc++;
    //float dinc=powf(1.1, (float) inc) * ddcu0[blockIdx.x];
    //dcu_cut= dinc*dinc;

    if(tid==0)
        while(Lfr_max < r_min)
        {
            Lfr_max=0;
            int j=1;
            start=0;
            for(int i=1; i<len; i++)
            {
                d = dist(x[i-1], x[i]);
                flag=0;
                if(dcu_cut>dcu0_cut)
                {
                    if(d<dcu_cut)
                    {
                        flag=1;
                    }
                }
		else if(resno[i] == (resno[i-1]+1)) //necessary??
		{
				if(d<dcu_cut)
				{
					flag=1;
				}
		}
                if(flag==1)
                {
                    j++;
                    if(i==(len-1))
                    {
                        if(j > Lfr_max)
                        {
                            Lfr_max=j;
                            start_max=start;
                            end_max=i;
                        }
                        j=1;
                    }
                }
                else
                {
                    if(j>Lfr_max)
                    {
                        Lfr_max=j;
                        start_max=start;
                        end_max=i-1;
                    }

                    j=1;
                    start=i;
                }
                if(Lfr_max >= r_min)
                {
                    start_max2=start_max;
                    end_max2=end_max;
                }
            }


            if(Lfr_max < r_min)
            {
                inc++;
                float dinc=powf(1.1, (float) inc) * ddcu0[blockIdx.x];
                dcu_cut= dinc*dinc;
            }

        }
    //if(tid==0)
    {
        //	cuPrintf("2-->   %d  %d   inc %d\n",start_max2,end_max2,inc);
    }
    int a=0x00000000;
    a=a|end_max2;
    a=a<<16;
    a=a|start_max2;

    return a;


}


__device__ void get_initial_fgt(
    float x[][3],
    float y[][3],
    int xresno[],
    int yresno[],
    int x_len,
    int y_len,
    int y2x2[],
    float *score)
{

    const int tid=threadIdx.y*blockIdx.x+threadIdx.x;
    int fra_min=4;
    int fra_min1=fra_min-1;

    int sd1,sd2;
    sd1=find_max_frag(x,xresno, x_len);
    sd2=find_max_frag(y,yresno,y_len);

    unsigned int s1=0x0000ffff;
    unsigned int e1=0xffff0000;
    unsigned int s2=0x0000ffff;
    unsigned int e2=0xffff0000;

    s1=s1&sd1;
    e1=e1&sd1;
    e1=e1>>16;

    s2=s2&sd2;
    e2=e2&sd2;
    e2=e2>>16;

    //if(tid==0)
    //	cuPrintf("===> %d  %d  %d   %d \n",s1,e1,s2,e2);

    int Lx = e1-s1+1;
    int Ly = e2-s2+1;
    int L_fr=(Lx<=Ly?Lx:Ly);

    /*
    	volatile __shared__ int ifr[l2];
    	if(Lx<Ly || (Lx==Ly && x_len<=y_len))
    	{
    		for(int i=tid; i<L_fr; i=i+32)
    		{
    			ifr[i]=s1+i;
    		}
    	}
    	else if(Lx>Ly || (Lx==Ly && x_len>y_len))
    	{
    		for(int i=tid; i<L_fr; i=i+32)
    		{
    			ifr[i]=s2+i;
    		}
    	}
    */

    int ifr=0;
    if(Lx<Ly || (Lx==Ly && x_len<=y_len))
    {
        ifr=s1;
    }
    else if(Lx>Ly || (Lx==Ly && x_len>y_len))
    {
        ifr=s2;
    }



    int nn1, nn2;
    int L0=(x_len<=y_len?x_len:y_len);
    /*
    if(L_fr==L0)
    {
    	nn1= (int)(L0*0.1);
    	nn2= (int)(L0*0.89);

    	int j=tid;
    	for(int i=tid+nn1; i<=nn2; i=i+32,j=j+32)
    	{
    		ifr[j]=ifr[i];
    	}
    	L_fr=nn2-nn1+1;

    }
    */

    if(L_fr==L0)
    {
        nn1= (int)(L0*0.1);
        nn2= (int)(L0*0.89);

        ifr=ifr+nn1;
        L_fr=nn2-nn1+1;

    }

    //if(tid==0)
    //	cuPrintf("L_fr %d %d %d\n",L_fr,ifr[0],ifr[L_fr-1]);

    float tmscore, tmscore_max=-1;

    int i, j, k;
    int k_best=tid;
    int min_len=0;
    if(Lx<Ly || (Lx==Ly && x_len<=y_len))
    {
        min_len=(L_fr<=y_len?L_fr: y_len);
    }
    else
    {
        min_len=(x_len<=L_fr?x_len:L_fr);
    }


    int min_ali= (int) (min_len/2.5);
    if(min_ali<=fra_min1)  min_ali=fra_min1;

    if(Lx<Ly || (Lx==Ly && x_len<=y_len))
    {
        nn1 = -y_len+min_ali;
        nn2 = L_fr-min_ali;
    }
    else
    {
        nn1 = -L_fr+min_ali;
        nn2 = x_len-min_ali;
    }

    //if(tid==0)
    //	cuPrintf("%d  %d  %d \n",L_fr,nn1,nn2);


    int  y2x_[l2];
    for(k=nn1+tid; k<=nn2; k=k+32)
    {
        for(j=0; j<y_len; j++)
            y2x_[j]=-1;

        if(Lx<Ly || (Lx==Ly && x_len<=y_len))
        {
            for(j=0; j<y_len; j++)
            {
                i=j+k;
                if(i>=0 && i<L_fr)
                {
                    y2x_[j]=ifr+i;
                }
            }
        }
        else
        {
            for(j=0; j<L_fr; j++)
            {
                i=j+k;
                if(i>=0 && i<x_len)
                {
                    y2x_[ifr+j]=i;
                }
            }

        }
        /*
        	if(k==-42)
                    {
                            for(int m=0;m<y_len;m++)
                                   	if(m<L_fr)
                                   	cuPrintf("%d %d\n",ifr[m],y2x_[m]);
                           		else
                                   	cuPrintf("   %d\n",y2x_[m]);
                    }
        */
        tmscore=get_score_fast(x, y, x_len, y_len, y2x_);
        if(tmscore>=tmscore_max)
        {
            tmscore_max=tmscore;
            k_best=k;
        }
    }


    volatile __shared__ float sscore[32];
    volatile __shared__  int  sscore_i[32];
    sscore_i[tid]=k_best;
    sscore[tid]=tmscore_max;

    if(tid<16)
    {
        if(sscore[tid]<sscore[tid+16])
        {
            sscore[tid]=sscore[tid+16];
            sscore_i[tid]=sscore_i[tid+16];
        }
    }

    if(tid<8)
    {
        if(sscore[tid]<sscore[tid+8])
        {
            sscore[tid]=sscore[tid+8];
            sscore_i[tid]=sscore_i[tid+8];
        }
    }
    if(tid<4)
    {
        if(sscore[tid]<sscore[tid+4])
        {
            sscore[tid]=sscore[tid+4];
            sscore_i[tid]=sscore_i[tid+4];
        }
    }
    if(tid<2)
    {
        if(sscore[tid]<sscore[tid+2])
        {
            sscore[tid]=sscore[tid+2];
            sscore_i[tid]=sscore_i[tid+2];
        }
    }
    if(tid<1)
    {
        if(sscore[tid]<sscore[tid+1])
        {
            sscore[tid]=sscore[tid+1];
            sscore_i[tid]=sscore_i[tid+1];
        }
    }

    k=sscore_i[0];

    //if(tid==0)
    //{
    //if(blockIdx.x<10)
    //	printf("0%d (%d %d) %d  %f\n",blockIdx.x,nn1,nn2,sscore_i[0],sscore[0]);
    //else
    //	printf("%d (%d %d) %d  %f\n",blockIdx.x,nn1,nn2,sscore_i[0],sscore[0]);
    //cuPrintf("1 k %d tm %f == 2 k %d  tm %f\n",sscore_i[0],sscore[0],sscore_i[1],sscore[1]);
    //}

    for(j=tid; j<y_len; j=j+32)
        y2x2[j]=-1;


    if(Lx<Ly || (Lx==Ly && x_len<=y_len))
    {
        for(j=tid; j<y_len; j=j+32)
        {
            i=j+k;
            if(i>=0 && i<L_fr)
            {
                y2x2[j]=ifr+i;
            }
        }
    }
    else
    {
        for(j=tid; j<L_fr; j=j+32)
        {
            i=j+k;
            if(i>=0 && i<x_len)
            {
                y2x2[ifr+j]=i;
            }
        }
    }
}
__global__ void get_initial_fgt2(
    float x[][3],
    float y[][3],
    int xresno[],
    int yresno[],
    int x_len,
    int y_len[],
    const int l22,
    float *s)
{
    //cuPrintf("== %d \n",y_len[blockIdx.x]);
    //if(blockIdx.x==7||blockIdx.x==8)
    {
        get_initial_fgt(
            x,
            &y[blockIdx.x*l22],
	    &xresno[blockIdx.x*l22],
    	    &yresno[blockIdx.x*l22],
            x_len,
            y_len[blockIdx.x],
            invmap[blockIdx.x],
            NULL);

        /*
        	const int tid=threadIdx.y*blockIdx.x+threadIdx.x;
        	int i=0;
        	for(i=tid;i<l2;i=i+32)
        	{
        		s[blockIdx.x*l2+i]=invmap[blockIdx.x][i];
        	}
        */
    }
}
