#include "hip/hip_runtime.h"
/*************************************************************************
	> File Name: init_and_end.cu
	> Author: DB
	> Mail: dongboyaojiayou@163.com
	> Created Time: Thu 23 Apr 2015 05:19:33 PM CST
 ************************************************************************/

__global__ void parameter_set4search(int xlen, int ylen[],const int n1)
{
    const int tid=blockIdx.x* blockDim.x * blockDim.y
                  +threadIdx.y* blockDim.x+threadIdx.x;
    if(tid<n1)
    {


        float dcu0=4.25;
        float d0;
        int Lnorm= xlen<=ylen[tid]?xlen:ylen[tid];
        if(Lnorm<=19)
        {
            d0=0.168;
        }
        else
        {
            d0=(1.24*pow((Lnorm*1.0-15), 1.0/3)-1.8);
        }
        float D0_MIN=d0+0.8;
        d0=D0_MIN;

        float d0_search=d0;
        if(d0_search>8) d0_search=8;
        if(d0_search<4.5) d0_search=4.5;
        float score_d8=1.5*pow(Lnorm*1.0, 0.3)+3.5;

        dLnorm[tid]=Lnorm;
        ddcu0[tid]=dcu0;
        dd0[tid]=d0;
        dD0_MIN[tid]=D0_MIN;
        dd0_search[tid]=d0_search;
        dscore_d8[tid]=score_d8;

        dtmscore[tid]=-1.0;
        dtmscore2[tid]=-1.0;
    }
    //cuPrintf(" %d %d %f %f %f %f %f\n",
    //tid,Lnorm,dcu0/10,d0/10,D0_MIN/10,d0_search/10,score_d8/10);
}


__global__ void copytocpu(float s1[],float s2[],const int n1)
{
    const int tid=blockIdx.x* blockDim.x * blockDim.y
                  +threadIdx.y* blockDim.x+threadIdx.x;
    if(tid<n1)
    {
        s1[tid]=dtmscore[tid];
        s2[tid]=dtmscore2[tid];
        //printf("%f \n",dtmscore[tid]);
    }
}
