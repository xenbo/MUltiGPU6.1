#include "hip/hip_runtime.h"
__device__ void NWDP_TM2(
    int secx[],
    int secy[],
    int len1,
    int len2,
    float gap_open,
    int j2i[],
    const int l,
    float val[],
    char path[]
)
{

    int i, j;
    float h, v, d;

    const int tid=threadIdx.y*blockDim.x+threadIdx.x;
    for(i=tid; i<=len1; i+=32)
    {
        val[i*(l+1)+0]=0;
        path[i*(l+1)+0]=0;
    }

    for(j=tid; j<=len2; j+=32)
    {
        val[j]=0;
        path[j]=0;
        j2i[j]=-1;
    }

    int nv=1;
    i=tid+1;
    j=1;

    while(i<=len1)
    {
        if(i<=nv)
        {
            if(j<=len2)
            {

                if(secx[i-1]==secy[j-1])
                {
                    d=val[(i-1)*(l+1)+j-1]+ 1.0;
                }
                else
                {
                    d=val[(i-1)*(l+1)+j-1];
                }


                h=val[(i-1)*(l+1)+j];
                if(path[(i-1)*(l+1)+j])
                    h += gap_open;

                v=val[i*(l+1)+j-1];
                if(path[i*(l+1)+j-1])
                    v += gap_open;


                if(d>=h && d>=v)
                {
                    path[i*(l+1)+j]=1;
                    val[i*(l+1)+j]=d;
                }
                else
                {
                    path[i*(l+1)+j]=0;
                    if(v>=h)
                        val[i*(l+1)+j]=v;
                    else
                        val[i*(l+1)+j]=h;
                }
                j++;
            }
        }
        nv++;
        if(j>len2)
        {
            i+=32;
            j=1;
        }
    }

    if(tid==0)
    {
        i=len1;
        j=len2;
        while(i>0 && j>0)
        {
            if(path[i*(l+1)+j])
            {
                j2i[j-1]=i-1;
                i--;
                j--;
            }
            else
            {
                h=val[(i-1)*(l+1)+j];
                if(path[(i-1)*(l+1)+j]) h +=gap_open;

                v=val[i*(l+1)+j-1];
                if(path[i*(l+1)+j-1]) v +=gap_open;

                if(v>=h)
                    j--;
                else
                    i--;
            }
        }
    }
}

__device__ void smooth(int *sec, int len)
{
    int i, j;
    //smooth single  --x-- => -----
    for(i=2; i<len-2; i++)
    {
        if(sec[i]==2 || sec[i]==4)
        {
            j=sec[i];
            if(sec[i-2] != j)
            {
                if(sec[i-1] != j)
                {
                    if(sec[i+1] != j)
                    {
                        if(sec[i+2] != j)
                        {
                            sec[i]=1;
                        }
                    }
                }
            }
        }
    }

    //   smooth float
    //   --xx-- => ------

    for(i=0; i<len-5; i++)
    {
        //helix
        if(sec[i] != 2)
        {
            if(sec[i+1] != 2)
            {
                if(sec[i+2] == 2)
                {
                    if(sec[i+3] == 2)
                    {
                        if(sec[i+4] != 2)
                        {
                            if(sec[i+5] != 2)
                            {
                                sec[i+2]=1;
                                sec[i+3]=1;
                            }
                        }
                    }
                }
            }
        }

        //beta
        if(sec[i] != 4)
        {
            if(sec[i+1] != 4)
            {
                if(sec[i+2] ==4)
                {
                    if(sec[i+3] == 4)
                    {
                        if(sec[i+4] != 4)
                        {
                            if(sec[i+5] != 4)
                            {
                                sec[i+2]=1;
                                sec[i+3]=1;
                            }
                        }
                    }
                }
            }
        }
    }

    //smooth connect
    for(i=0; i<len-2; i++)
    {
        if(sec[i] == 2)
        {
            if(sec[i+1] != 2)
            {
                if(sec[i+2] == 2)
                {
                    sec[i+1]=2;
                }
            }
        }
        else if(sec[i] == 4)
        {
            if(sec[i+1] != 4)
            {
                if(sec[i+2] == 4)
                {
                    sec[i+1]=4;
                }
            }
        }
    }

}

__device__ int sec_str(float dis13, float dis14,
                       float dis15, float dis24,
                       float dis25, float dis35)
{
    int s=1;

    float delta=2.1;
    if(fabs(dis15-6.37)<delta)
    {
        if(fabs(dis14-5.18)<delta)
        {
            if(fabs(dis25-5.18)<delta)
            {
                if(fabs(dis13-5.45)<delta)
                {
                    if(fabs(dis24-5.45)<delta)
                    {
                        if(fabs(dis35-5.45)<delta)
                        {
                            s=2;
                            return s;
                        }
                    }
                }
            }
        }
    }

    delta=1.42;
    if(fabs(dis15-13)<delta)
    {
        if(fabs(dis14-10.4)<delta)
        {
            if(fabs(dis25-10.4)<delta)
            {
                if(fabs(dis13-6.1)<delta)
                {
                    if(fabs(dis24-6.1)<delta)
                    {
                        if(fabs(dis35-6.1)<delta)
                        {
                            s=4; //strand
                            return s;
                        }
                    }
                }
            }
        }
    }

    if(dis15 < 8)
    {
        s=3; //turn
    }


    return s;
}


__device__ void make_sec(float x[][3], int len, int sec[])
{

    const int tid=threadIdx.y*blockDim.x+threadIdx.x;
    for(int i=tid; i<len; i=i+32)
    {
        sec[i]=-1;
    }


    int j1, j2, j3, j4, j5;
    float d13, d14, d15, d24, d25, d35;
    for(int i=tid; i<len; i+=32)
    {
        sec[i]=1;
        j1=i-2;
        j2=i-1;
        j3=i;
        j4=i+1;
        j5=i+2;

        if(j1>=0 && j5<len)
        {
            d13=sqrt(dist(x[j1], x[j3]));
            d14=sqrt(dist(x[j1], x[j4]));
            d15=sqrt(dist(x[j1], x[j5]));
            d24=sqrt(dist(x[j2], x[j4]));
            d25=sqrt(dist(x[j2], x[j5]));
            d35=sqrt(dist(x[j3], x[j5]));
            sec[i]=sec_str(d13, d14, d15, d24, d25, d35);
        }

    }
    if(tid==0)
        smooth(sec, len);
}

__device__  void get_initial_ss(
    float x[][3],
    float y[][3],
    int secx[],
    int secy[],
    int x_len,
    int y_len,
    int *y2x,
    const int l,
    float val[],
    char path[])
{

    make_sec(x, x_len, secx);
    make_sec(y, y_len, secy);

    float gap_open=-1.0;
    NWDP_TM2(secx,
             secy,
             x_len,
             y_len,
             gap_open,
             y2x,
             l,
             val,
             path);

}

__global__ void get_initial_ss2(
    float x[][3],
    float y[][3],
    int x_len,
    int y_len[],
    const int l22,
    float val[],
    char path[],
    float *s)
{

    get_initial_ss(	x,
                    &y[blockIdx.x*l22],
                    secx[blockIdx.x],
                    secy[blockIdx.x],
                    x_len,
                    y_len[blockIdx.x],
                    invmap[blockIdx.x],l22,
                    &val[blockIdx.x*(x_len+1)*(l22+1)],
                    &path[blockIdx.x*(x_len+1)*(l22+1)]);


    /*
    	const int tid=threadIdx.y*blockDim.x+threadIdx.x;
    	for(int i=tid;i<l2;i=i+32)
    	{
    		s[blockIdx.x*l2+i]=invmap[blockIdx.x][i];
    	}
    */
}


