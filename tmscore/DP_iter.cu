#include "hip/hip_runtime.h"
__device__ void DP_iter(
    float x[][3],
    float y[][3],
    int x_len,
    int y_len,
    int g1,
    int g2,
    int iteration_max,
    const int l,
    float val[],
    char path[],
    float *s
)
{
    float tmscore=0.0,gap_open[2]= {-0.6, 0};
    int iteration=0, i, j, k;
    float  tmscore_max=0, tmscore_old=0;
    const int tid=threadIdx.y*blockDim.x+threadIdx.x;

    int score_sum_method=8, simplify_step=40;
    volatile __shared__ int sk;
    float d0=dd0[blockIdx.x];
    float d02=d0*d0;

    /*
       if(tid==0)
    	cuPrintf("%f %f %f \n",
    			t1[blockIdx.x][0],
    			t1[blockIdx.x][1],
    			t1[blockIdx.x][2]);

    */

    int g=g1;
    for(; g<g2; g++)
    {
        for(iteration=0; iteration<iteration_max; iteration++)
        {

            DNW(x, y, x_len, y_len,
                t1[blockIdx.x],
                u1[blockIdx.x],
                d02,
                gap_open[g],
                invmap[blockIdx.x],
                l,
                val,
                path);

            k=0;
            if(tid<3)
            {
                for(j=0; j<y_len; j++)
                {
                    i=invmap[blockIdx.x][j];
                    if(i>=0&&i<x_len) //aligned
                    {
                        xtm1[blockIdx.x][k][tid]=x[i][tid];
                        ytm1[blockIdx.x][k][tid]=y[j][tid];
                        k++;
                        //if(tid==0 && iteration==2)
                        //	cuPrintf("%f ->%f\n",y[j][2],x[i][2]);
                    }
                }
                if(tid==0)
                    sk=k;
            }
            //****************************************************
            /*
              if(tid==0)
             {
             	cuPrintf("T: %f  %f  %f  %d  %d  sk:%d tmscore:%f\n",
             			t1[blockIdx.x][0],
             			t1[blockIdx.x][1],
             			t1[blockIdx.x][2],
             			x_len,y_len,sk,tmscore);
             }
            */
            //****************************************************
            //if(tid==0)
            //	printf("  %f  %f  %f  -- TM ylen  %d  %d %f \n",t1[blockIdx.x][0],t1[blockIdx.x][1],t1[blockIdx.x][2],y_len,k,tmscore);

            tmscore= TMscore8_search(
                         xtm1[blockIdx.x],
                         ytm1[blockIdx.x],
                         sk,
                         t1[blockIdx.x],
                         u1[blockIdx.x],
                         simplify_step,
                         score_sum_method,1);


            if(tmscore>tmscore_max)
            {
                tmscore_max=tmscore;

                for(int i=tid; i<y_len; i=i+32)
                {
                    invmap2[blockIdx.x][0][i]=invmap[blockIdx.x][i];
                }
            }
            if(fabs(tmscore_old-tmscore)<0.000001)
            {
                break;
            }
            tmscore_old=tmscore;
        }// for iteration

    }//for gapopen

    if(tid==0)
    {
        dtmscore2[blockIdx.x]=tmscore_max;
        //cuPrintf("========== %d %f\n", sk,tmscore_max);
    }
}



__global__ void GP_iter(
    float x[][3],
    float y[][3],
    int x_len,
    int y_len[],
    int g1,
    int g2,
    int iteration_max,
    const int l22,
    float val[],
    char path[],
    float *s)
{
    DP_iter(x,
            &y[(blockIdx.x)*l22],
            x_len,
            y_len[blockIdx.x],
            g1,g2,iteration_max,l22,
            &val[blockIdx.x*(x_len+1)*(l22+1)],
            &path[blockIdx.x*(x_len+1)*(l22+1)],
            NULL);



    const int tid=threadIdx.y*blockDim.x+threadIdx.x;
    volatile __shared__  int flag;
    if(tid==0)
    {   flag=0;
        if(dtmscore[blockIdx.x]<dtmscore2[blockIdx.x])
            flag=1;
    }

    if(flag)
    {   for(int i=tid; i<l22; i=i+32)
        {
            // invmap[blockIdx.x][i]=invmap2[blockIdx.x][0][i];
            //  s[blockIdx.x*l2+i]=invmap2[blockIdx.x][0][i];
        }
    }
}


__global__ void GP_iter1(
    float x[][3],
    float y[][3],
    int x_len,
    int y_len[],
    int g1,
    int g2,
    int iteration_max,
    const int l22,
    float val[],
    char path[],
    float *s)
{
    DP_iter(x,
            &y[(blockIdx.x)*l22],
            x_len,
            y_len[blockIdx.x],
            g1,g2,iteration_max,l22,
            &val[blockIdx.x*(x_len+1)*(l22+1)],
            &path[blockIdx.x*(x_len+1)*(l22+1)],
            NULL);


    const int tid=threadIdx.y*blockDim.x+threadIdx.x;
    volatile __shared__  int flag;
    if(tid==0)
    {   flag=0;
        if(dtmscore[blockIdx.x]<dtmscore2[blockIdx.x])
        {
            dtmscore[blockIdx.x]=dtmscore2[blockIdx.x];
            flag=1;
        }
    }

    if(flag)
    {   for(int i=tid; i<l22; i=i+32)
        {
            invmapbak[blockIdx.x][i]=invmap2[blockIdx.x][0][i];
            //s[blockIdx.x*l2+i]=invmap2[blockIdx.x][8][i];
        }
    }
}

__global__ void GP_iter2(
    float x[][3],
    float y[][3],
    int x_len,
    int y_len[],
    int g1,
    int g2,
    int iteration_max,
    const int l22,
    float val[],
    char path[],
    float *s)
{


    const int tid=threadIdx.y*blockDim.x+threadIdx.x;
    volatile __shared__  int flag;

    //if(blockIdx.x==0)
    {

        if(tid==0)
            if(dtmscore2[blockIdx.x]>dtmscore[blockIdx.x]*0.2)
                flag=1;

        if(flag)
        {
            DP_iter(x,
                    &y[(blockIdx.x)*l22],
                    x_len,
                    y_len[blockIdx.x],
                    g1,g2,iteration_max,l22,
                    &val[blockIdx.x*(x_len+1)*(l22+1)],
                    &path[blockIdx.x*(x_len+1)*(l22+1)],
                    NULL);
            if(tid==0)
            {
                flag=0;
                if(dtmscore[blockIdx.x]<dtmscore2[blockIdx.x])
                {
                    dtmscore[blockIdx.x]=dtmscore2[blockIdx.x];
                    flag=1;
                }
            }
            if(flag)
            {
                for(int i=tid; i<l22; i=i+32)
                {
                    invmapbak[blockIdx.x][i]=invmap2[blockIdx.x][0][i];
                    //s[blockIdx.x*l2+i]=invmapbak[blockIdx.x][i];
                }
            }
        }
    }
}

__global__ void GP_iter3(
    float x[][3],
    float y[][3],
    int x_len,
    int y_len[],
    int g1,
    int g2,
    int iteration_max,
    const int l22,
    float val[],
    char path[],
    float *s)
{


    const int tid=threadIdx.y*blockDim.x+threadIdx.x;
    volatile __shared__  int flag;

    float  ddcc=0.4;
    if(dLnorm[blockIdx.x] <= 40) ddcc=0.1;
    if(tid==0)
        if(dtmscore2[blockIdx.x]>dtmscore[blockIdx.x]*ddcc)
            flag=1;

    if(flag)
    {
        DP_iter(x,
                &y[(blockIdx.x)*l22],
                x_len,
                y_len[blockIdx.x],
                g1,g2,iteration_max,l22,
                &val[blockIdx.x*(x_len+1)*(l22+1)],
                &path[blockIdx.x*(x_len+1)*(l22+1)],
                NULL);
        if(tid==0)
        {
            flag=0;
            if(dtmscore[blockIdx.x]<dtmscore2[blockIdx.x])
            {
                dtmscore[blockIdx.x]=dtmscore2[blockIdx.x];
                flag=1;
            }
        }
        if(flag)
        {
            for(int i=tid; i<l22; i=i+32)
            {
                invmapbak[blockIdx.x][i]=invmap2[blockIdx.x][0][i];
                // s[blockIdx.x*l2+i]=invmapbak[blockIdx.x][i];
            }
        }
    }
}


__global__ void GP_iter4(
    float x[][3],
    float y[][3],
    int x_len,
    int y_len[],
    int g1,
    int g2,
    int iteration_max,
    const int l22,
    float val[],
    char path[],
    float *s)
{


    const int tid=threadIdx.y*blockDim.x+threadIdx.x;
    volatile __shared__  int flag;

    float  ddcc=0.4;
    if(dLnorm[blockIdx.x] <= 40) ddcc=0.1;
    if(tid==0)
        if(dtmscore2[blockIdx.x]>dtmscore[blockIdx.x]*ddcc)
            flag=1;

    if(flag)
    {
        DP_iter(x,
                &y[(blockIdx.x)*l22],
                x_len,
                y_len[blockIdx.x],
                g1,g2,iteration_max,l22,
                &val[blockIdx.x*(x_len+1)*(l22+1)],
                &path[blockIdx.x*(x_len+1)*(l22+1)],
                NULL);
        if(tid==0)
        {
            flag=0;
            if(dtmscore[blockIdx.x]<dtmscore2[blockIdx.x])
            {
                dtmscore[blockIdx.x]=dtmscore2[blockIdx.x];
                flag=1;
            }
        }
        if(flag)
        {
            for(int i=tid; i<l22; i=i+32)
            {
                invmapbak[blockIdx.x][i]=invmap2[blockIdx.x][0][i];
                //s[blockIdx.x*l2+i]=invmapbak[blockIdx.x][i];
            }
        }
    }

    /////////////////////////test

    //	for(int i=tid;i<l22;i=i+32)
    //		         s[blockIdx.x*l2+i]=invmapbak[blockIdx.x][i];


}

__global__ void GP_iter5(
    float x[][3],
    float y[][3],
    int x_len,
    int y_len[],
    int g1,
    int g2,
    int iteration_max,
    const int l22,
    float val[],
    char path[],
    float *s)
{


    const int tid=threadIdx.y*blockDim.x+threadIdx.x;
    volatile __shared__  int flag;

    float  ddcc=0.4;
    if(dLnorm[blockIdx.x] <= 40) ddcc=0.1;
    if(tid==0)
        if(dtmscore2[blockIdx.x]>dtmscore[blockIdx.x]*ddcc)
            flag=1;

    if(flag)
    {
        DP_iter(x,
                &y[(blockIdx.x)*l22],
                x_len,
                y_len[blockIdx.x],
                g1,g2,iteration_max,l22,
                &val[blockIdx.x*(x_len+1)*(l22+1)],
                &path[blockIdx.x*(x_len+1)*(l22+1)],
                NULL);
        if(tid==0)
        {
            flag=0;
            if(dtmscore[blockIdx.x]<dtmscore2[blockIdx.x])
            {
                dtmscore[blockIdx.x]=dtmscore2[blockIdx.x];
                flag=1;
            }
        }
        if(flag)
        {
            for(int i=tid; i<l22; i=i+32)
            {
                invmapbak[blockIdx.x][i]=invmap2[blockIdx.x][0][i];
                //s[blockIdx.x*l2+i]=invmapbak[blockIdx.x][i];
            }
        }
    }

    /////////////////////////test

    //for(int i=tid;i<l22;i=i+32)
    //			         s[blockIdx.x*l2+i]=invmapbak[blockIdx.x][i];


}

