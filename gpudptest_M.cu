#include <sstream>
#include <iostream>
#include <fstream>
#include <omp.h>
#include <cmath>
#include <cstdlib>
#include <ctime>
#include <cstdio>
#include <cstring>
#include"tmscore/cuPrintf.cuh"
#include"tmscore/cuPrintf.cu"
#define nmax 360	///每个大段分配额576个
#define ss_j 16
#define ss_b 200

//#define l1 180
//#define l2 280
const int l1=500;
const int l2=280;
using namespace std;
struct pdblist
{
    int flag_len;	//段长 90，100，110，120，130......
    float *xyz;
    int *resno;
    int *len;	//pdb 长度
    int *id;	//pdb id号
    int num;	//pdb 个数
};

const int flag_len[ss_j]= {90,100,110,120,130,140,150,160,170,180,200,220,240,260,280,1200}; //16个大段
struct pdblist *plist;
const int NGPU=2;

__device__ int dLnorm[nmax];
__device__ float ddcu0[nmax];
__device__ float dd0[nmax];
__device__ float dD0_MIN[nmax];
__device__ float dd0_search[nmax];
__device__ float dscore_d8[nmax];

__device__ float xtm1[nmax][l2][3],ytm1[nmax][l2][3];
__device__ float u1[nmax][3][3],t1[nmax][3];
//__device__ float val[nmax][l1+1][l2+1];
//__device__ char path[nmax][l1+1][l2+1];
//__device__ float score[nmax][l1+1][l2+1];
__device__ int secx[nmax][l1];
__device__ int secy[nmax][l2];

__device__ float dtmscore[nmax];
__device__ float dtmscore2[nmax];

__device__ int  invmap[nmax][l2+2];
__device__ int invmapbak[nmax][l2+2];
__device__ int invmap2[nmax][8][l2+2];

#include"tmscore/basic.cu"
#include"tmscore/init_and_end.cu"
#include"tmscore/Kabsch.cu"
#include"tmscore/dnw.cu"
#include"tmscore/TMscore8_search2.cu"
#include"tmscore/TMscore8_search3.cu"
#include"tmscore/get_init.cu"
#include"tmscore/get_initial_ss.cu"
#include"tmscore/get_initial_local.cu"
#include"tmscore/get_initial_ssplus.cu"
#include"tmscore/get_initial_fgt.cu"
#include"tmscore/final.cu"
#include"tmscore/DP_iter.cu"


int switch_l(int l)
{
    if (l<=90) {
        return  0;
    }
    if (l<=100) {
        return  1;
    }
    if (l<=110) {
        return  2;
    }
    if (l<=120) {
        return  3;
    }
    if (l<=130) {
        return  4;
    }
    if (l<=140) {
        return  5;
    }
    if (l<=150) {
        return  6;
    }
    if (l<=160) {
        return  7;
    }
    if (l<=170) {
        return  8;
    }
    if (l<=180) {
        return  9;
    }
    if (l<=200) {
        return  10;
    }
    if (l<=220) {
        return  11;
    }
    if (l<=240) {
        return  12;
    }
    if (l<=260) {
        return  13;
    }
    if (l<=280) {
        return  14;
    }
    return 15;
}

void get_xyz(string line, float *x, float *y, float *z, int *no)
{
    char cstr[50];

    strcpy(cstr, (line.substr(30, 8)).c_str());
    sscanf(cstr, "%f", x);

    strcpy(cstr, (line.substr(38, 8)).c_str());
    sscanf(cstr, "%f", y);

    strcpy(cstr, (line.substr(46, 8)).c_str());
    sscanf(cstr, "%f", z);

    strcpy(cstr, (line.substr(22, 4)).c_str());
    sscanf(cstr, "%d", no);
}

int read_PDB(const char *filename, float a[][3], int *resno)
{
    int i=0;
    string line, str;
    string atom ("ATOM ");

    ifstream fin (filename);
    if (fin.is_open())
    {
        while ( fin.good() )
        {
            getline(fin, line);
            if(line.compare(0, atom.length(), atom)==0)
            {
                if( line.compare(12, 4, "CA  ")==0 ||\
                        line.compare(12, 4, " CA ")==0 ||\
                        line.compare(12, 4, "  CA")==0 )
                {
                    if( line.compare(16, 1, " ")==0 ||\
                            line.compare(16, 1, "A")==0 )
                    {
                        get_xyz(line, &a[i][0], &a[i][1], &a[i][2],&resno[i]);
                        i++;
                    }
                }
            }
        }
        fin.close();
    }
    return i;
}


int read_list(char *filelist)
{
    float a[2000][3];
    int b[2000];
    int id=0;
    ifstream fin(filelist);
    if (fin.is_open())
    {
        while (fin.good())
        {   memset(&a[0][0],0,2000*3*sizeof(float));

            string line, str;
            getline(fin, line);
            if(line.length()>0)
            {
                string s=line.substr(0,line.length());
                int l=read_PDB(s.c_str(), a, b); //装进a[][]
                int n=switch_l(l);
                while(plist[n].num>=nmax)	n=n+ss_j;
                memcpy(plist[n].xyz+(plist[n].num*plist[n].flag_len*3),&a[0][0],l*3*sizeof(float));
		memcpy(plist[n].resno+(plist[n].num*plist[n].flag_len),b,l*sizeof(int));
                plist[n].len[plist[n].num]=l;
                plist[n].id[plist[n].num]=id;
                plist[n].num++;
                id++;
            }
        }
        fin.close();
    }

    return id;
}


int main(int argv,char *argc[])
{
    //******************************************************************************************
    //******************************加载pdb 数据*************************************************
    //******************************************************************************************
    float *hscore1[ss_b];
    float *hscore2[ss_b];
    plist=(struct pdblist *)malloc(sizeof(struct pdblist)*ss_b);
    for(int i=0; i<ss_b; i++)
    {
        plist[i].xyz=(float*)malloc(sizeof(float)*nmax*flag_len[i%ss_j]*3);
	plist[i].resno=(int *)malloc(sizeof(int)*nmax*flag_len[i%ss_j]);
        plist[i].id=(int *)malloc(sizeof(int)*nmax);
        plist[i].len=(int *)malloc(sizeof(int)*nmax);
        memset(plist[i].xyz,0,sizeof(float)*nmax*flag_len[i%ss_j]*3);
	memset(plist[i].resno,0,sizeof(int)*nmax*flag_len[i%ss_j]);
	memset(plist[i].len,0,sizeof(int)*nmax);
        memset(plist[i].id,0,sizeof(int)*nmax);
        plist[i].flag_len=flag_len[i%ss_j];
        plist[i].num=0;
        hscore1[i]=(float*)malloc(sizeof(float)*nmax);
        hscore2[i]=(float*)malloc(sizeof(float)*nmax);
    }

    char *p="name.txt";
    const int n2=read_list(p);
    float p1[2000][3];
    int b1[2000];
    const int len1=read_PDB("1.pdb",p1,b1);

    printf("******************pdb1 %d *********************\n",len1);
    printf("******************pdb0 %d *********************\n",n2);

    //********************************************************************************************
    //******************************gpu处理 数据***************************************************
    //********************************************************************************************
    dim3 b_(32,1);
    dim3 b_2(4,8);

    hipStream_t *streams=(hipStream_t*)malloc(sizeof(hipStream_t)*NGPU);
    double t1 = omp_get_wtime();
    #pragma omp parallel num_threads(NGPU)
    {
        int ngid = omp_get_thread_num();
        hipSetDevice(ngid);
        {
            hipStreamCreate(&(streams[ngid]));
            float *dp1,(*dp21)[3];
            float *dp0,(*dp20)[3];

            float *score1,*score2;
            int *dln,*dresno0,*dresno1;
            float *valline1;
            char  *pathline1;
            float  *dscore1;
            hipMalloc((void**)&(dp1),sizeof(float)*(len1+3)*3);
            hipMalloc((void**)&(dp0),sizeof(float)*l2*3*nmax);
	    dp21=(float (*)[3])&(dp1[0]);
            dp20=(float (*)[3])&(dp0[0]);


            hipMalloc((void**)&dln,sizeof(int)*nmax);
	    hipMalloc((void**)&dresno1,sizeof(int)*l1);
	    hipMalloc((void**)&dresno0,sizeof(int)*nmax*l2);
            hipMalloc((void**)&(score1),sizeof(float)*nmax);
            hipMalloc((void**)&(score2),sizeof(float)*nmax);
            
            hipMalloc((void**)&(valline1),sizeof(float)*nmax*(len1+1)*(l2+1));
            hipMalloc((void**)&(pathline1),sizeof(char)*nmax*(len1+1)*(l2+1));
            hipMalloc((void**)&(dscore1),sizeof(float)*nmax*(len1+1)*(l2+1));

            hipMemcpyAsync(dp1,&p1[0][0],sizeof(float)*(len1*3),hipMemcpyHostToDevice,streams[ngid]);//加载目标pdb1数据
	    hipMemcpyAsync(dresno1,b1,sizeof(int)*(len1),hipMemcpyHostToDevice,streams[ngid]);//加载目标pdb1数据
            //cudaPrintfInit();
            for(int c1=ngid; c1<ss_b; c1=c1+NGPU)//openmp 可以NGPU个批次计算
            {
                if(plist[c1].num>0)
                {
                    const int n1=plist[c1].num;
                    const int ss=plist[c1].flag_len;

                    dim3 g(n1,1);
                    //******************************数据加载到 gpu***************************
                    hipMemcpyAsync(dp0,plist[c1].xyz,sizeof(float)*(nmax*ss*3),hipMemcpyHostToDevice,streams[ngid]);
                    hipMemcpyAsync(dln,plist[c1].len,sizeof(int)*(nmax),hipMemcpyHostToDevice,streams[ngid]);
		    hipMemcpyAsync(dresno0,plist[c1].resno,sizeof(int)*nmax*ss,hipMemcpyHostToDevice,streams[ngid]);

                    //******************************gpu处理 设置参数************************
                    parameter_set4search<<<g,b_,0,streams[ngid]>>>(len1,dln,n1);
                    //******************************gpu处理 数据1***************************
                    get_initial2<<<g,b_,0,streams[ngid]>>>(dp21,dp20,len1,dln,ss,score1);
                    detailed_search1<<<g,b_2,0,streams[ngid]>>>(dp21,dp20,len1,dln,40,8,ss,score1);
                    GP_iter1<<<g,b_2,0,streams[ngid]>>>(dp21,dp20,len1,dln,0,2,30,ss,valline1,pathline1,score1);

                    //******************************gpu处理 数据2***************************
                    get_initial_ss2<<<g,b_,0,streams[ngid]>>>(dp21,dp20,len1,dln,ss,valline1,pathline1,score1);
                    detailed_search2<<<g,b_2,0,streams[ngid]>>>(dp21,dp20,len1,dln,40,8,ss,score1);
                    GP_iter2<<<g,b_2,0,streams[ngid]>>>(dp21,dp20,len1,dln,0,2,30,ss,valline1,pathline1,score1);

                    //******************************gpu处理 数据3***************************
                    get_initial_local2<<<g,b_,0,streams[ngid]>>>(dp21,dp20,len1,dln,ss,valline1,pathline1,score1);
                    detailed_search3<<<g,b_2,0,streams[ngid]>>>(dp21,dp20,len1,dln,40,8,ss,score1);
                    GP_iter3<<<g,b_2,0,streams[ngid]>>>(dp21,dp20,len1,dln,0,2,2,ss,valline1,pathline1,score1);

                    //******************************gpu处理 数据4***************************
                    get_initial_ssplus2<<<g,b_,0,streams[ngid]>>>(dp21,dp20,len1,dln,ss,dscore1,valline1,pathline1,score1);
                    detailed_search4<<<g,b_2,0,streams[ngid]>>>(dp21,dp20,len1,dln,40,8,ss,score1);
                    GP_iter4<<<g,b_2,0,streams[ngid]>>>(dp21,dp20,len1,dln,0,2,30,ss,valline1,pathline1,score1);

                    //******************************gpu处理 数据5***************************
                    get_initial_fgt2<<<g,b_,0,streams[ngid]>>>(dp21,dp20,dresno1,dresno0,len1,dln,ss,score1);
                    detailed_search5<<<g,b_2,0,streams[ngid]>>>(dp21,dp20,len1,dln,40,8,ss,score1);
                    GP_iter5<<<g,b_2,0,streams[ngid]>>>(dp21,dp20,len1,dln,1,2,2,ss,valline1,pathline1,score1);

                    //******************************gpu处理 数据6***************************
	            detailed_search6<<<g,b_2,0,streams[ngid]>>>(dp21,dp20,len1,dln,1,8,ss,score1);
	            Gfinal_TMscore8_search<<<g,b_2,0,streams[ngid]>>>(dp21,dp20,len1,dln,ss,score1);
                    
		    //******************************gpu 数据拷回***************************
                    copytocpu<<<g,b_2,0,streams[ngid]>>>(score1,score2,n1);
                    hipMemcpyAsync(hscore1[c1],score1,sizeof(float)*(n1),hipMemcpyDeviceToHost,streams[ngid]);
                    hipMemcpyAsync(hscore2[c1],score2,sizeof(float)*(n1),hipMemcpyDeviceToHost,streams[ngid]);


                }
            }
            //cudaPrintfDisplay(stdout,true);
            //cudaPrintfEnd();
            //hipDeviceSynchronize();
            //****************************** 数据 释放**********************************
            hipFree(dp0);
            hipFree(dp1);
            hipFree(score1);
            hipStreamDestroy(streams[ngid]);
        }

    }



    double t2 = omp_get_wtime();
    /*
    	for(int i=1; i<=(n1/32*32)*l2; i++)
        {
            printf("%.0f ",hscore1[i-1]);
    		if(i%l2==0)printf("\n\n");
        }
     */
    //********************************************************************************************
    //******************************数据  输出***************************************************
    //********************************************************************************************
    char c0[6];
    for(int i=0; i<ss_b; i++)
    {
        if(plist[i].num>0)
        {
            for(int j=0; j<plist[i].num; j++)
            {
                int id=plist[i].id[j];
                printf("%s%d  %.6f  %.6f\n",get50(id,c0),id,hscore1[i][j],hscore2[i][j]);
            }
        }
        free(hscore1[i]);
        free(hscore2[i]);
    }

    printf("time: %f ms\n",t2-t1);
    hipDeviceReset();
}
